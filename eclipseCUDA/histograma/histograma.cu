#include <stdio.h>
#include <time.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

#define NUMELEMENTSV 33554432;
#define NUMELEMENTSH 8;
#define THREADSPERBLOCK 1024;
#define LOOPS 10;
/**
 * CUDA Kernel Device code
 * Calcula el histograma de un vector pasado. 
 * Versión directa con operaciones atómicas a vector H en memoria de video
 */
__global__ void histogram(int *V, int * H, int numElementsV, int numElementsH)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
	
if (i < numElementsV)
  {
	int index = V[i] % numElementsH;
	atomicAdd((H +index), 1); 
  }
	
}


/**
 * CUDA Kernel Device code
 * Calcula el histograma de un vector pasado.
 * Versión directa con operaciones atómicas a memoria compartida por bloque y un solo acceso atómico final a la memoria global de la CPU, tras sincronización de los hilos
 * con el resultado a memorida de video.
 */
 __global__ void
 histogramShared(int *V, int * H, int numElementsV, int numElementsH)
 {
 __shared__ int acc[8];  ///tamaño máximo del vector H.Ver si se puede optimizar para  crear en tiempo de ejecución el vector dentro del kernel.
 ///además, no se como inicializar la variable a cero inicialmente.

  int i = blockDim.x * blockIdx.x + threadIdx.x;
   
  if (i % blockDim.x == 0)  ///si es el hilo del principio de un bloque, me encargaré de inicializar a cero la variable contador y después de ser el hilo que escriba a memoria global de la GPU.
    for (int j=0; j < 8; j++)
      acc[j] = 0;

     __syncthreads();    

 if (i < numElementsV)
   {
     int index = V[i] % numElementsH;
     atomicAdd((&acc[0] + index), 1); 
     
   }
   __syncthreads();    

   if (i % blockDim.x == 0)    
     for (int j = 0; j < numElementsH; j++)     
        atomicAdd((H + j), acc[j]); 
 
   
 }

/**
 * CUDA Kernel Device code
 * Suma por reducción de los elementos de un vector
 */
 __global__ void
 sumHistogram(int * h, int blocksPerGrid) ///blocksPerGrid dice el número de bloques  que realmente habría, o sea, los reales multiplicados por 2.
 {
     ///blockDim equivale al tamño del histograma, ya que se creó el bloque con tantos hilos como tamaño del histograma.

     int i = blockDim.x * blockIdx.x + threadIdx.x; ///posicion dentro del vector del histograma.
     int j = (blocksPerGrid -1 - blockIdx.x) * blockDim.x + threadIdx.x; ///valor  que voy a sumar dentro del histograma  que le corresponde. 
     ///O sea, el primer bloque(equivalente al tamaño  de elementos del histograma) se sumará con el último, el segundo, con el penúltimo.....

    h[i] = h[i] + h[j];
    
 }

/**
 * CUDA Kernel Device code
 * Calcula el histograma de un vector pasado.
 * Versión directa con operaciones atómicas a memoria de video, pero con un histograma por bloque.
 * Se sigue utilizando un vector H, pero su tamaño es numElementsH * número de bloques.
 * O sea, hay un histograma de tamaño numElementsH por cada bloque y su ubicación en memoria es como vector, un histograma seguido del otro
 */
__global__ void
histogramByBlock(int *V, int * H, int numElementsV, int numElementsH)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
	
if (i < numElementsV)
  {	
	int index = (V[i] % numElementsH) + (blockIdx.x * numElementsH); ///me posiciono en histograma asociado a este bloque y en la posición correspondiente
	atomicAdd((H +index), 1); 
  }
	
}

/**
 * CUDA Kernel Device code
 * Calcula el histograma de un vector pasado.
 * Versión directa con operaciones atómicas a memoria de video, pero con un histograma por bloque.
 * Se sigue utilizando un vector H, pero su tamaño es numElementsH * número de bloques.
 * O sea, hay un histograma de tamaño numElementsH por cada bloque y su ubicación en memoria es como vector, un histograma seguido del otro
 */
 __global__ void
 histogramByBlockShared(int *V, int * H, int numElementsV, int numElementsH)
 {
   __shared__ int acc[8];  ///tamaño máximo del vector H.Ver si se puede optimizar para  crear en tiempo de ejecución el vector dentro del kernel.
   ///además, no se como inicializar la variable a cero inicialmente.
  
    int i = blockDim.x * blockIdx.x + threadIdx.x;
     
    if (i % blockDim.x == 0)  ///si es el hilo del principio de un bloque, me encargaré de inicializar a cero la variable contador y después de ser el hilo que escriba a memoria global de la GPU.
      for (int j=0; j < 8; j++)
        acc[j] = 0;
  
       __syncthreads();    
  
     
     int index = V[i] % numElementsH;
    
     if (i < numElementsV)                  
        atomicAdd((&acc[0] + index), 1); 
            
     __syncthreads();    
  
     if (i % blockDim.x == 0)    
     {
        int index2 = blockIdx.x * numElementsH; ///me posiciono en histograma asociado a este bloque y en la posición correspondiente
        for (int j = 0; j < numElementsH; j++)     
        atomicAdd((H + index2 + j), acc[j]); 

     }


 }
/**
 * CUDA Kernel Device code
 * Calcula el histograma de un vector pasado mediante diferentes métodos.
 * Devuelve un puntero hacia el el vector del histograma ya calculado
 */
int * calculateHistogramByGpu(int * vector,int numElementsV, int numElementsH, bool byBlock, int threadsPerBlock, bool shared)
{
	size_t sizeH,sizeV;
	int *h_H = NULL;
	int *d_V = NULL;
	int *d_H = NULL;	
    hipError_t err = hipSuccess;
	
	sizeV =  numElementsV * sizeof(int);

	if (threadsPerBlock >1024)
	    threadsPerBlock = 1024;  ///para no sobrepasar el límite de bloque. Realmente este valor se deberá de obtener de la función CUDA adecuada, puesto que podría no ser
							    		
    int blocksPerGrid = (numElementsV + threadsPerBlock - 1) / threadsPerBlock;
	
 if (byBlock)
	///si creo la versión de un histograma por bloque	
	sizeH = numElementsH * blocksPerGrid * sizeof(int) ;
 else
   ///si creo la versión de un histograma único para todo el vector.   
	sizeH = numElementsH * sizeof(int);

 	h_H = (int *)malloc(sizeH);
	
    // Verify that allocations succeeded
    if (h_H == NULL)
    {
        fprintf(stderr, "Failed to allocate host vector Histograma!\n");
        exit(EXIT_FAILURE);
    }

    // Initialize the host input vector H a cero
    printf("\nVector Histrograma Inicializado con 0:");
    for (int i = 0; i < (sizeH / sizeof(int)); ++i)    
		h_H[i] = 0;
	
        
    // Allocate the device vector V       
    err = hipMalloc((void **)&d_V, sizeV);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector V (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

// Allocate the device vector H    

    err = hipMalloc((void **)&d_H, sizeH);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector H (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    // Copy the host input vectors V in host memory to the device input vectors in
    // device memory
    //printf("\nCopy input data from the host memory vector V to the CUDA device");
    err = hipMemcpy(d_V, vector, sizeV, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector V from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the host input vectors H in host memory to the device input vectors in
    // device memory
    //printf("\nCopy input data from the host memory vector H to the CUDA device");
    err = hipMemcpy(d_H, h_H, sizeH, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector H from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

	// Launch the Vector Add CUDA Kernel   
	if (byBlock)	
	{
        if (shared)
        {

            printf("\nCUDA kernel - histogramByBlockShared - launch with %d blocks of %d threads", blocksPerGrid, threadsPerBlock);       
            histogramByBlockShared<<<blocksPerGrid, threadsPerBlock>>>(d_V, d_H, numElementsV, numElementsH);
            
            if ((blocksPerGrid % 2) !=  0) //si los bloques no son pares, 
                printf("\nBloques impares, todavía no implementada solución, puede fallar");       
    
            while (blocksPerGrid > 1)
            {
                
                sumHistogram<<<blocksPerGrid /2 , numElementsH>>>(d_H, blocksPerGrid);
                blocksPerGrid /= 2;
                
            }
    
        }
        else
        {

            printf("\nCUDA kernel - histogramByBlock - launch with %d blocks of %d threads", blocksPerGrid, threadsPerBlock);       
            histogramByBlock<<<blocksPerGrid, threadsPerBlock>>>(d_V, d_H, numElementsV, numElementsH);
            
            if ((blocksPerGrid % 2) !=  0) //si los bloques no son pares, 
                printf("\nBloques impares, todavía no implementada solución, puede fallar");       
    
            while (blocksPerGrid > 1)
            {
                
                sumHistogram<<<blocksPerGrid /2 , numElementsH>>>(d_H, blocksPerGrid);
                blocksPerGrid /= 2;
                
            }
    
        }
        
            

	}
	else
	{
        if (shared)
        {
            printf("\nCUDA kernel -histogramShared- launch with %d blocks of %d threads", blocksPerGrid, threadsPerBlock);       
            histogramShared<<<blocksPerGrid, threadsPerBlock>>>(d_V, d_H, numElementsV, numElementsH);        
        }
        else 
        {
            printf("\nCUDA kernel -histogram- launch with %d blocks of %d threads", blocksPerGrid, threadsPerBlock);       
            histogram<<<blocksPerGrid, threadsPerBlock>>>(d_V, d_H, numElementsV, numElementsH);        
    
        }
	
	}
    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
	
    // Copy the device result vector V in device memory to the host result vector
    // in host memory.
    //printf("\nCopy output data from the CUDA device vector V to the host memory");
    err = hipMemcpy(vector, d_V, sizeV, hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector V from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    // Copy the device result vector H in device memory to the host result vector
    // in host memory.
    //printf("\nCopy output data from the CUDA device vector H to the host memory");
    err = hipMemcpy(h_H, d_H, sizeH, hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector H from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    // Free device global memory
    err = hipFree(d_V);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector V (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free device global memory
    err = hipFree(d_H);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector V (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


return h_H;
}


/**
 * Calcula el histograma de un vector pasado.

 */
int * calculateHistogramByCpu(int * vector, int numElementsV, int numElementsH)
{
////Version calculo histograma por CPU.     

int * h_H = (int *)malloc(numElementsH * sizeof(int));
  
// Verify that allocations succeeded
if (h_H == NULL)
{
    fprintf(stderr, "Failed to allocate host vector Histograma!\n");
    exit(EXIT_FAILURE);
}

unsigned t0,t1;

t0 = clock();

///Calculo el tiempo que tardaría si se hiciera por CPU
   for (int i = 0; i < numElementsV; i++)
      h_H[vector[i] % numElementsH] = h_H[vector[i] % numElementsH] +1;
     
 t1 = clock();
      double time =  (double (t1-t0)/CLOCKS_PER_SEC);
    
  
///Show Vector H
printf("\nTiempo empleado en calculo por CPU :  %f segundos",time);

return h_H ;
}

/**
 * Host main routine
 */
int main(void)
{

    int * h_H = NULL; ///puntero hacia el vector Histograma.
    hipError_t err = hipSuccess;

    // Print the vector length to be used, and compute its size
    int numElementsV = NUMELEMENTSV;
	int numElementsH = NUMELEMENTSH;
    int threadsPerBlock = THREADSPERBLOCK;
    int repeatLoop = LOOPS;
	    
    printf("\nVector V de %d elementos", numElementsV);
    // Allocate the host input vector V
    int *h_V = (int *)malloc(numElementsV * sizeof(int));

    // Verify that allocations succeeded
    if (h_V == NULL)
    {
        fprintf(stderr, "Failed to allocate host vector V!\n");
        exit(EXIT_FAILURE);
    }

    // Initialize the host input vector V
   // printf("\nVector V Inicializado con :");

    for (int i = 0; i < numElementsV; ++i)    	 
		h_V[i] = rand() % 1000000;  ///número aleatorio entre 0 y 1000000
	 		//h_V[i] = i;

     
     

//*******************************************************************************************************************************   
////Version calculo histograma por CPU.     
//*******************************************************************************************************************************   

for (int r = 0; r < repeatLoop; r++)
{
     h_H = calculateHistogramByCpu(h_V, numElementsV, numElementsH);
    
    if (r == repeatLoop-1)  ///si es la última iteración, muestro los resultados
    {
         ///Show Vector H
        printf("\nResultado Vector Histograma Calculado por CPU ");
        for (int i = 0; i < numElementsH; ++i)    
            printf("\n[%d]", h_H[i]);

    } 
      
    free(h_H);    ///la función realiza la solicitud de memoria y debemos borrarla después de utilizarla antes de volver a llamar a la función
}
    
                    
    

    
    
//*******************************************************************************************************************************   
////Version calculo histograma por GPU. Versión, todos los hilos de todos los bloques a un mismo vector de histograma.
////Con acceso directo a memoria global de la GPU con escrituras atómicas.
//*******************************************************************************************************************************   
for (int r = 0; r < repeatLoop; r++)
{   
    h_H = calculateHistogramByGpu(h_V, numElementsV, numElementsH, false, threadsPerBlock,false);
    if (r == repeatLoop-1)  ///si es la última iteración, muestro los resultados
    {
         ///Show Vector H         
        printf("\nResultado Vector Histograma versión Sin bloque y Sin memoria compartida :");        
        for (int i = 0; i < numElementsH; ++i)    
            printf("\n[%d]", h_H[i]);

    } 
    free(h_H);
}


//*******************************************************************************************************************************   
////Version calculo histograma por GPU. Versión, todos los hilos de todos los bloques a un mismo vector de histograma.
////Con acceso directo a compartida por bloque mediante escrituras atómicas  y después solo el primer hilo de cada bloque escribirá, incrementando de forma
/// atómica los resultandos en la memoria global de la GPU. Al final son más escrituras atómicas, son las mismas que en la versión sin memoria compartida, más
/// una escritura, de todo el vector histograma, por bloque. Aún así, es tanta la diferencia de velocidad entre memoria global y compartida, que se mejora
///muchísimo el rendimiento.l
//*******************************************************************************************************************************   

for (int r = 0; r < repeatLoop; r++)
{   
    h_H = calculateHistogramByGpu(h_V, numElementsV, numElementsH, false, threadsPerBlock,true);

    if (r == repeatLoop-1)  ///si es la última iteración, muestro los resultados
    {
         ///Show Vector H                 
        printf("\nResultado Vector Histograma Versión Sin bloque y Memroria compartida :");
        for (int i = 0; i < numElementsH; ++i)    
            printf("\n[%d]", h_H[i]);

    } 

    free(h_H);
}


//*******************************************************************************************************************************   
////Version calculo histograma por GPU. Versión, todos los hilos de un mismo bloque, escribirán de forma atómica hacia
/// su propio vector histograma, de manera que las escrituras se reparten entre  los hilos de un histograma por bloque. Este vector histograma,
////realmente es uno por cada bloque, pero que se asignan como un único vector de tamaño "numero elementos vector histograma" * "numero de bloques a ejecutar"
/// Así que el acceso al histograma del primer bloque será la dirección del vector, la del segundo bloque, la dirección del vector + tamaño del numero de elemntos,(normalmente 8)
////Este algoritmo mejora, ya que parece hacer uso de los accesos a memoria mejorados de las GPU, ya sea por la optimización de escrituras/lecturas de palabras grandes
///como posibles paralelismos en las operaciones de lectura/escritura, también puede mejorar el hecho de que las operaciones atómicas, solo deben de protegerse de los hilos de un mismo bloque
/// y no de todos los bloques en ejecución.
///Este método requiere de la ejecución de dos kernel, el segundo es el encargado de sumar todos los histogramas generados por cada bloque y devolver 
// uno solo, que es colocado en las primeras posiciones del vector histograma general.
///La mejora es sustancial.
//*******************************************************************************************************************************   

for (int r = 0; r < repeatLoop; r++)
{   
    h_H = calculateHistogramByGpu(h_V, numElementsV, numElementsH, true, threadsPerBlock,false);   
    if (r == repeatLoop-1)  ///si es la última iteración, muestro los resultados
    {
         ///Show Vector H                 
         printf("\nResultado Vector Histograma Con bloques y Sin Memoria Compartida :");        
        for (int i = 0; i < numElementsH; ++i)    
            printf("\n[%d]", h_H[i]);

    } 
    free(h_H);
}



//*******************************************************************************************************************************   
////Version calculo histograma por GPU. Versión, todos los hilos de un mismo bloque, escribirán de forma atómica hacia
/// su propio vector histograma almacenado en memoria compartida para cada bloque. De manera que las escrituras se reparten entre  los hilos de un histograma por bloque.
/// Al final, solo el hilo cero de cada bloque, es el que se encarga de volcar su vector de memoria compartida (equivalente a las escrituras atómicas de todos los hilos de un bloque)
///  a la memoria global de la GPU. Este traspaso se realiza también mediante operaciones atómicas.
///Este método requiere de la ejecución de dos kernel, el segundo es el encargado de sumar todos los histogramas generados por cada bloque y devolver 
// uno solo, que es colocado en las primeras posiciones del vector histograma general.

//*******************************************************************************************************************************   

for (int r = 0; r < repeatLoop; r++)
{   
    h_H = calculateHistogramByGpu(h_V, numElementsV, numElementsH, true, threadsPerBlock,true);   
    if (r == repeatLoop-1)  ///si es la última iteración, muestro los resultados
    {
         ///Show Vector H                          
         printf("\nResultado Vector Histograma Con bloques y Memoria Compartida :");
        for (int i = 0; i < numElementsH; ++i)    
            printf("\n[%d]", h_H[i]);

    } 
    free(h_H);

}

    // Free host memory
    free(h_V);
	

    // Reset the device and exit
    err = hipDeviceReset();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("\nDone\n");
    return 0;
}

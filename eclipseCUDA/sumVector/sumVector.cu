#include <stdio.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

///variables globales para el device
__device__ float d_suma;

/**
 * CUDA Kernel Device code
 * Incrementa cada componente del vector A con numElements elementos
 */
__global__ void
sumVector(float *A, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        ///d_suma = d_suma + A[i]; tarda 1.345504 y las sumas nunca dan lo mismo
    	atomicAdd(&d_suma, A[i]); ///tarda 116.553  pero la suma siempre da lo mismo.

    }
}



/**
 * Host main routine
 */
int main(void)
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Print the vector length to be used, and compute its size
    int numElements = 50000000;
    size_t size = numElements * sizeof(float);
    printf("[Vector increment of %d elements]\n", numElements);

    // Allocate the host input vector A
    float *h_A = (float *)malloc(size);


    // Verify that allocations succeeded
    if (h_A == NULL)
    {
        fprintf(stderr, "Failed to allocate host vector!\n");
        exit(EXIT_FAILURE);
    }

    // Initialize the host input vector
    printf("\nVector Inicializado con :\n");

    for (int i = 0; i < numElements; ++i)
    {
    		h_A[i] = i;
        	//printf("[%f]", h_A[i]);
    }
    // Allocate the device vector A
    float *d_A = NULL;
    err = hipMalloc((void **)&d_A, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the host input vectors A in host memory to the device input vectors in
    // device memory
    printf("\nCopy input data from the host memory to the CUDA device\n");
    err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 1024;
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);

    hipEvent_t start,stop;
    float time_ms;
    float suma = 0;
    hipMemcpyToSymbol(HIP_SYMBOL(d_suma), &suma, sizeof(float));


    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start,0);

    sumVector<<<blocksPerGrid, threadsPerBlock>>>(d_A, numElements);
    err = hipGetLastError();

    hipEventRecord(stop,0);

    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_ms, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    printf("\nTime elapsed by kernel: %f\n", time_ms);

    printf("\nValor de la suma del vector : %f",suma);

    hipMemcpyFromSymbol(&suma,HIP_SYMBOL(d_suma),sizeof(float));

    printf("\nValor de la suma del vector : %f",suma);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the device result vector in device memory to the host result vector
    // in host memory.
    printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector A from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    // Free device global memory
    err = hipFree(d_A);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free host memory
    free(h_A);

    // Reset the device and exit
    err = hipDeviceReset();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("Done\n");
    return 0;
}

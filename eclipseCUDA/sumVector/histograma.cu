#include <stdio.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>


/**
 * CUDA Kernel Device code
 * Calcula el histograma de un vector pasado. 
 * Versión directa con operaciones atómicas a vector H en memoria de video
 */
__global__ void
histogram(int *V, int * H, int numElementsV, int numElementsH)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
	
if (i < numElementsV)
  {
	int index = V[i] % numElementsH;
	atomicAdd((H +index), 1); 
  }
	
}

/**
 * CUDA Kernel Device code
 * Calcula el histograma de un vector pasado.
 * Versión directa con operaciones atómicas a memoria de video, pero con un histograma por bloque.
 * Se sigue utilizando un vector H, pero su tamaño es numElementsH * número de bloques.
 * O sea, hay un histograma de tamaño numElementsH por cada bloque y su ubicación en memoria es como vector, un histograma seguido del otro
 */
__global__ void
histogramByBlock(int *V, int * H, int numElementsV, int numElementsH)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
	
if (i < numElementsV)
  {	
	int index = (V[i] % numElementsH) + (blockIdx.x * numElementsH); ///me posiciono en histograma asociado a este bloque y en la posición correspondiente
	atomicAdd((H +index), 1); 
  }
	
}

/**
 * CUDA Kernel Device code
 * Calcula el histograma de un vector pasado.
 * Versión directa con operaciones atómicas a memoria compartida por un bloque y un solo acceso atómico final, tras sincronización de los hilos
 * con el resultado.
 * a memorida de video.
 */
__global__ void
histogramShared(int *V, int * H, int numElementsV, int numElementsH)
{
__shared__ int acc[50];  ///tamaño máximo del vector H.Ver si se puede optimizar para  crear en tiempo de ejecución el vector dentro del kernel.

 int i = blockDim.x * blockIdx.x + threadIdx.x;
	
if (i < numElementsV)
  {
	int index = V[i] % numElementsH;
	atomicAdd((H +index), 1); 
  }
	
}

/**
 * CUDA Kernel Device code
 * Calcula el histograma de un vector pasado.
 * Versión directa con operaciones atómicas a memoria compartida por un bloque y un solo acceso atómico final, tras sincronización de los hilos
 * con el resultado.
 * a memorida de video.
 * Devuelve un puntero hacia el el vector del histograma ya calculado
 */
int * calculateHistogramByGpu(int * vector,int numElementsV, int numElementsH, bool byBlock, int threadsPerBlock)
{
	size_t sizeH,sizeV;
	int *h_H = NULL;
	int *d_V = NULL;
	int *d_H = NULL;	
    hipError_t err = hipSuccess;
	
	sizeV =  numElementsV * sizeof(int);

	if (threadsPerBlock >1024)
	    threadsPerBlock = 1024;  ///para no sobrepasar el límite de bloque. Realmente este valor se deberá de obtener de la función CUDA adecuada, puesto que podría no ser
							    		
    int blocksPerGrid = (numElementsV + threadsPerBlock - 1) / threadsPerBlock;
	
 if (byBlock)
	///si creo la versión de un histograma por bloque	
	sizeH = numElementsH * blocksPerGrid * sizeof(int) ;
 else
   ///si creo la versión de un histograma único para todo el vector.   
	sizeH = numElementsH * sizeof(int);

 	h_H = (int *)malloc(sizeH);
	
    // Verify that allocations succeeded
    if (h_H == NULL)
    {
        fprintf(stderr, "Failed to allocate host vector Histograma!\n");
        exit(EXIT_FAILURE);
    }

    // Initialize the host input vector H a cero
    printf("\nVector Histrograma Inicializado con 0:");
    for (int i = 0; i < (sizeH / sizeof(int)); ++i)    
		h_H[i] = 0;
	
        
    // Allocate the device vector V       
    err = hipMalloc((void **)&d_V, sizeV);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector V (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

// Allocate the device vector H    

    err = hipMalloc((void **)&d_H, sizeH);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector H (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    // Copy the host input vectors V in host memory to the device input vectors in
    // device memory
    printf("\nCopy input data from the host memory vector V to the CUDA device");
    err = hipMemcpy(d_V, vector, sizeV, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector V from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the host input vectors H in host memory to the device input vectors in
    // device memory
    printf("\nCopy input data from the host memory vector H to the CUDA device");
    err = hipMemcpy(d_H, h_H, sizeH, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector H from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

	// Launch the Vector Add CUDA Kernel   
	if (byBlock)	
	{
	    printf("\nCUDA kernel - histogramByBlock - launch with %d blocks of %d threads", blocksPerGrid, threadsPerBlock);       
		histogramByBlock<<<blocksPerGrid, threadsPerBlock>>>(d_V, d_H, numElementsV, numElementsH);

	}
	else
	{
		printf("\nCUDA kernel -histogram- launch with %d blocks of %d threads", blocksPerGrid, threadsPerBlock);       
		histogram<<<blocksPerGrid, threadsPerBlock>>>(d_V, d_H, numElementsV, numElementsH);
	
	}
    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
	
    // Copy the device result vector V in device memory to the host result vector
    // in host memory.
    printf("\nCopy output data from the CUDA device vector V to the host memory");
    err = hipMemcpy(vector, d_V, sizeV, hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector V from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    // Copy the device result vector H in device memory to the host result vector
    // in host memory.
    printf("\nCopy output data from the CUDA device vector H to the host memory");
    err = hipMemcpy(h_H, d_H, sizeH, hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector H from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    // Free device global memory
    err = hipFree(d_V);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector V (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free device global memory
    err = hipFree(d_H);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector V (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


return h_H;
}

/**
 * Calcula el histograma de un vector pasado.
 * Versión directa con operaciones atómicas a memoria compartida por un bloque y un solo acceso atómico final, tras sincronización de los hilos
 * con el resultado.
 * a memorida de video.
 */
/*int * calculateHistogramByCpu(int * vector, int sizeHistogram, bool byBlock)
{
int numElementsH = 8;	
return 
}*/

/**
 * Host main routine
 */
int main(void)
{

	hipError_t err = hipSuccess;

    // Print the vector length to be used, and compute its size
    int numElementsV = 50000000;
	int numElementsH = 8;
	int threadsPerBlock = 1024;
	    
    printf("\nVector V de %d elementos", numElementsV);
    // Allocate the host input vector V
    int *h_V = (int *)malloc(numElementsV * sizeof(int));

    // Verify that allocations succeeded
    if (h_V == NULL)
    {
        fprintf(stderr, "Failed to allocate host vector V!\n");
        exit(EXIT_FAILURE);
    }

    // Initialize the host input vector V
    printf("\nVector V Inicializado con :");

    for (int i = 0; i < numElementsV; ++i)    
	 {
		//	h_V[i] = rand();  ///(float)RAND_MAX;
			h_V[i] = i;
		if  (numElementsV<1025)
			printf("\n[%d]", h_V[i]); ///solo muestro por pantalla si es menor o igual de 1024
     }	
	 

    int * h_H = calculateHistogramByGpu(h_V, numElementsV, numElementsH, false, threadsPerBlock);

///Show Vector H
    printf("\nResultado Vector Histograma  :");
    for (int i = 0; i < numElementsH; ++i)    
		 {			
			printf("\n[%d]", h_H[i]);
		}	

free(h_H);

     h_H = calculateHistogramByGpu(h_V, numElementsV, numElementsH, true, threadsPerBlock);

///Show Vector H
    printf("\nResultado Vector Histograma  :");
    for (int i = 0; i < numElementsH; ++i)    
		 {			
			printf("\n[%d]", h_H[i]);
		}	

free(h_H);





    // Free host memory
    free(h_V);
	

    // Reset the device and exit
    err = hipDeviceReset();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("\nDone\n");
    return 0;
}
